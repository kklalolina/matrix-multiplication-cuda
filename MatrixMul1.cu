#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

 /**
  * Matrix multiplication: C = A * B.
  * Host code.
  *
  * This sample implements matrix multiplication which makes use of shared memory
  * to ensure data reuse, the matrix multiplication is done using tiling approach.
  * It has been written for clarity of exposition to illustrate various CUDA programming
  * principles, not with the goal of providing the most performant generic kernel for matrix multiplication.
  * See also:
  * V. Volkov and J. Demmel, "Benchmarking GPUs to tune dense linear algebra,"
  * in Proc. 2008 ACM/IEEE Conf. on Supercomputing (SC '08),
  * Piscataway, NJ: IEEE Press, 2008, pp. Art. 31:1-11.
  */

  // System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#define K 2
#define L 3

template <int BLOCK_SIZE>
__global__ void MatrixMulCUDA(float* C, float* A, float* B, int wA, int wB) {
    int bx = blockIdx.x;
    int by = blockIdx.y;

    int tx = threadIdx.x;
    int ty = threadIdx.y;
 
    int aBegin = wA * BLOCK_SIZE * by * K;

    int aEnd = aBegin + wA - 1;

    int aStep = BLOCK_SIZE;

    int bBegin = BLOCK_SIZE * bx * L;


    int bStep = BLOCK_SIZE * wB;

    float Csub[K * L] = { 0.0f };


    for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {

        __shared__ float As[K * BLOCK_SIZE][BLOCK_SIZE];

        __shared__ float Bs[BLOCK_SIZE][L * BLOCK_SIZE];


        for (int i = 0; i < K; ++i) {
            As[ty + BLOCK_SIZE * i][tx] = A[a + wA * (ty + BLOCK_SIZE * i) + tx];
        }
        for (int i = 0; i < L; ++i) {
            Bs[ty][tx + BLOCK_SIZE * i] = B[b + wB * ty + tx + BLOCK_SIZE * i];
        }
       
        __syncthreads();

      

#pragma unroll

       for (int k = 0;k < K;k++) {
            for (int l = 0;l < L;l++) {
                for (int i = 0;i < BLOCK_SIZE;i++) {
                    Csub[k * L + l] += As[ty + BLOCK_SIZE * k][i] * Bs[i][tx + BLOCK_SIZE * l];

                }
            }
        }

        __syncthreads();
    }

    int cstart = aBegin + bBegin + wB * ty + tx;
    for (int k = 0; k < K; ++k) {
        for (int l = 0; l < L; ++l) {
            C[cstart + (k * wB * BLOCK_SIZE) + l * BLOCK_SIZE] = Csub[k * L + l];
        }
    }
    

    
}

void ConstantInit(float* data, int size, float val) {
    for (int i = 0; i < size; ++i) {
        data[i] = val;
    }
}
/**
 * Run a simple test of matrix multiplication using CUDA
 */
int MatrixMultiply(int argc, char** argv,
    int block_size, const dim3& dimsA,
    const dim3& dimsB) {
    // Allocate host memory for matrices A and B
    unsigned int size_A = dimsA.x * dimsA.y;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float* h_A;
    checkCudaErrors(hipHostMalloc(&h_A, mem_size_A));
    unsigned int size_B = dimsB.x * dimsB.y;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float* h_B;
    checkCudaErrors(hipHostMalloc(&h_B, mem_size_B));
    hipStream_t stream;

    // Initialize host memory
    const float valB = 0.1f;
    ConstantInit(h_A, size_A, 1.0f);
    ConstantInit(h_B, size_B, valB);

    // Allocate device memory
    float* d_A, * d_B, * d_C;

    // Allocate host matrix C
    dim3 dimsC(dimsB.x, dimsA.y, 1);
    unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(float);
    float* h_C;
    checkCudaErrors(hipHostMalloc(&h_C, mem_size_C));

    if (h_C == NULL) {
        fprintf(stderr, "Failed to allocate host matrix C!\n");
        exit(EXIT_FAILURE);
    }

    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_A), mem_size_A));
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_B), mem_size_B));
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_C), mem_size_C));
    // Allocate CUDA events that we'll use for timing
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

    // copy host memory to device
    checkCudaErrors(
        hipMemcpyAsync(d_A, h_A, mem_size_A, hipMemcpyHostToDevice, stream));
    checkCudaErrors(
        hipMemcpyAsync(d_B, h_B, mem_size_B, hipMemcpyHostToDevice, stream));

    // Setup execution parameters
    dim3 threads(block_size, block_size);
    dim3 grid(dimsB.x / (threads.x * L), dimsA.x / (threads.y * K));
    // Create and start timer
    printf("Computing result using CUDA Kernel...\n");

    
    // Performs warmup operation using matrixMul CUDA kernel
    /*if (block_size == 16) {
        MatrixMulCUDA<16>
            << <grid, threads, 0, stream >> > (d_C, d_A, d_B, dimsA.x, dimsB.x);
    }
    else {
        MatrixMulCUDA<32>
            << <grid, threads, 0, stream >> > (d_C, d_A, d_B, dimsA.x, dimsB.x);
    }*/

    printf("done\n");
    checkCudaErrors(hipStreamSynchronize(stream));

    // Record the start event
    checkCudaErrors(hipEventRecord(start, stream));

    // Execute the kernel
    int nIter = 1;

    for (int j = 0; j < nIter; j++) {
        if (block_size == 16) {
            MatrixMulCUDA<16>
                << <grid, threads, 0, stream >> > (d_C, d_A, d_B, dimsA.x, dimsB.x);
        }
        else {
            MatrixMulCUDA<32>
                << <grid, threads, 0, stream >> > (d_C, d_A, d_B, dimsA.x, dimsB.x);
        }
    }
  
    // Record the stop event
    checkCudaErrors(hipEventRecord(stop, stream));

    // Wait for the stop event to complete
    checkCudaErrors(hipEventSynchronize(stop));

    float msecTotal = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

    // Compute and print the performance
    float msecPerMatrixMul = msecTotal / nIter;
    double flopsPerMatrixMul = 2.0 * static_cast<double>(dimsA.x) *
        static_cast<double>(dimsA.y) *
        static_cast<double>(dimsB.x);
    double gigaFlops =
        (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
    printf(
        "Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,"
        " WorkgroupSize= %u threads/block\n",
        gigaFlops, msecPerMatrixMul, flopsPerMatrixMul, threads.x * threads.y);

    // Copy result from device to host
    checkCudaErrors(
        hipMemcpyAsync(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost, stream));
    checkCudaErrors(hipStreamSynchronize(stream));

    printf("Checking computed result for correctness: ");
    bool correct = true;

    // test relative error by the formula
    //     |<x, y>_cpu - <x,y>_gpu|/<|x|, |y|>  < eps
    double eps = 1.e-6;  // machine zero
    
    /*printf("\n");
    for (int i = 0;i < dimsA.y;i++) {
        for (int j = 0;j < dimsA.y;j++) {
            printf("%.2f\t ", h_C[j + i * dimsA.y]);
        }
        printf("\n");
    }*/
    
    for (int i = 0; i < static_cast<int>(dimsC.x * dimsC.y); i++) {
        double abs_err = fabs(h_C[i] - (dimsA.x * valB));
        double dot_length = dimsA.x;
        double abs_val = fabs(h_C[i]);
        double rel_err = abs_err / abs_val / dot_length;

        if (rel_err > eps) {
            printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",
                i, h_C[i], dimsA.x * valB, eps);
            correct = false;
        }
    }

    printf("%s\n", correct ? "Result = PASS" : "Result = FAIL");

    // Clean up memory
    checkCudaErrors(hipHostFree(h_A));
    checkCudaErrors(hipHostFree(h_B));
    checkCudaErrors(hipHostFree(h_C));
    checkCudaErrors(hipFree(d_A));
    checkCudaErrors(hipFree(d_B));
    checkCudaErrors(hipFree(d_C));
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    printf(
        "\nNOTE: The CUDA Samples are not meant for performance "
        "measurements. Results may vary when GPU Boost is enabled.\n");

    if (correct) {
        return EXIT_SUCCESS;
    }
    else {
        return EXIT_FAILURE;
    }
}


/**
 * Program main
 */
int main(int argc, char** argv) {
    printf("[Matrix Multiply Using CUDA] - Starting...\n");

    if (checkCmdLineFlag(argc, (const char**)argv, "help") ||
        checkCmdLineFlag(argc, (const char**)argv, "?")) {
        printf("Usage -device=n (n >= 0 for deviceID)\n");
        printf("      -wA=WidthA -hA=HeightA (Width x Height of Matrix A)\n");
        printf("      -wB=WidthB -hB=HeightB (Width x Height of Matrix B)\n");
        printf("  Note: Outer matrix dimensions of A & B matrices" \
            " must be equal.\n");

        exit(EXIT_SUCCESS);
    }

    // This will pick the best possible CUDA capable device, otherwise
    // override the device ID based on input provided at the command line
    int dev = findCudaDevice(argc, (const char**)argv);

    const int block_size = 32;

    const int matrix_size = block_size * K * L * 5;
 
    dim3 dimsA(matrix_size, matrix_size, 1);
    dim3 dimsB(matrix_size, matrix_size, 1);


    // width of Matrix A
    if (checkCmdLineFlag(argc, (const char**)argv, "wA")) {
        dimsA.x = getCmdLineArgumentInt(argc, (const char**)argv, "wA");
    }

    // height of Matrix A
    if (checkCmdLineFlag(argc, (const char**)argv, "hA")) {
        dimsA.y = getCmdLineArgumentInt(argc, (const char**)argv, "hA");
    }

    // width of Matrix B
    if (checkCmdLineFlag(argc, (const char**)argv, "wB")) {
        dimsB.x = getCmdLineArgumentInt(argc, (const char**)argv, "wB");
    }

    // height of Matrix B
    if (checkCmdLineFlag(argc, (const char**)argv, "hB")) {
        dimsB.y = getCmdLineArgumentInt(argc, (const char**)argv, "hB");
    }

    if (dimsA.x != dimsB.y) {
        printf("Error: outer matrix dimensions must be equal. (%d != %d)\n",
            dimsA.x, dimsB.y);
        exit(EXIT_FAILURE);
    }

    printf("MatrixA(%d,%d), MatrixB(%d,%d)\n", dimsA.x, dimsA.y,
        dimsB.x, dimsB.y);

    checkCudaErrors(hipProfilerStart());
    int matrix_result = MatrixMultiply(argc, argv, block_size, dimsA, dimsB);
    checkCudaErrors(hipProfilerStop());

    exit(matrix_result);
}
